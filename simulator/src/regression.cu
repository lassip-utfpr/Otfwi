#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <math.h>
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include "regression.h"

#define xzt(x,z,t) ((x) + (z)*(X) + ((t)%4)*(X)*(Z))
#define xzt2(x,z,t) ((x) + (z)*(X) + ((t)%4)*(X)*(Z))
#define inbounds(x,z,offset) (((x)>=(offset) && (z)>=(offset) && (x)<(X)-(offset) && (z)<(Z)-(offset)))

//precisam ser iguais
#define WARP_SIZE (32)
#define BLOCK_SIZE (32)

#define prec_deriv (4)
#include "deriv_macros.h"

typedef void (*adj_func)(float*, float*);


const dim3 threadGrid(BLOCK_SIZE, BLOCK_SIZE);

float *P, *cquad, *source, *record_buffer, *initial, *recording_h;
float *P_ub, *P_uf, *grad, *observed, *adj_source, *grad_h, *simulated_h, *adj_source_h;
float *P_uf_full;
int *pos_source_x, *pos_source_z, *pos_sensor_x, *pos_sensor_z;
int X, Z, T, n_source, n_sensor;
int allocated = 0;
unsigned int n_blocksX, n_blocksZ, n_blocksS, n_blocksF;
dim3 blockGrid;



__global__ void
simulateFrame (float* P, float *cquad, int X, int Z, int t)
{
    //coordenadas no bloco
    const int x = threadIdx.x;
    const int z = threadIdx.y;

    //coordenadas em Ps (shared)
    const int x_s = threadIdx.x + prec_deriv;
    const int z_s = threadIdx.y + prec_deriv;

    //coordenadas da origem do bloco
    const int x_b = blockIdx.x * blockDim.x;
    const int z_b = blockIdx.y * blockDim.y;

    //coordenadas em P (global)
    const int x_g = x_b + x;
    const int z_g = z_b + z;

    const int shared_width = BLOCK_SIZE + 2*prec_deriv;
    const int tam_shared = (shared_width)*(shared_width);

    //nao vale a pena colocar cquad e P(t-2) na memoria shared de acordo com os testes
    __shared__ float Ps[shared_width][shared_width];


    if(z==0) //first warp in block
    {
	//copy P to shared memory
	for(int id=x; id<tam_shared; id+=WARP_SIZE)
	{
	    //coordenada dentro de Ps (shared) sendo lida de P
	    const int xx = id/shared_width;
	    const int zz = id%shared_width;

	    //coordenada correspondente na memoria global
	    const int x_c = x_b - prec_deriv + xx;
	    const int z_c = z_b - prec_deriv + zz;

	    //retirar if adicionando o anel de zeros
	    if(inbounds(x_c, z_c, prec_deriv))
		Ps[zz][xx] = P[xzt(x_c, z_c, t-1)];
	    else
		Ps[zz][xx] = 0.0f;
	}
    }
    __syncthreads();
    

    float lap = deriv_x(Ps, x_s, z_s) + deriv_z(Ps, x_s, z_s);

    //assume que nao tem anel de zeros forcados por indice, entao usa as bordas como anel
    if(inbounds(x_g, z_g, prec_deriv))
	P[xzt(x_g, z_g, t)] = -P[xzt(x_g, z_g, t - 2)] + 2 * Ps[z_s][x_s] + cquad[xzt (x_g, z_g, 0)] * lap;
    else
	P[xzt(x_g,z_g,t)] = 0.0f;
}


__global__ void
somaFonte(float *P, int X, int Z, int T, int t, int *pos_source_x, int *pos_source_z, float *source, int n_source, int flip, int idx=-1)
{
    const int indexF = threadIdx.x + blockDim.x*threadIdx.y + blockIdx.x*(BLOCK_SIZE*BLOCK_SIZE);
    const int n = indexF; 

    if(n>=n_source || (idx!=-1 && idx!=n))
	return;

    float fonte;
    if(flip)
	fonte = source[n * T + T - 1 - t];
    else
	fonte = source[n*T + t];

    P[xzt(pos_source_x[n], pos_source_z[n], t)] += fonte;
}


__global__ void
gravaBufferSensores2ordem(float *P, float *recording, int X, int Z, int T, int t, int *pos_sensor_x, int *pos_sensor_z, int n_sensor)
{
    //coordenadas no bloco
    const int indexS = threadIdx.x + blockDim.x*threadIdx.y + blockIdx.x*(BLOCK_SIZE*BLOCK_SIZE);
    const int n = indexS; 
    if(n<n_sensor)
	recording[n*T + t] = P[xzt2(pos_sensor_x[n], pos_sensor_z[n], t)];
}


__global__ void
calcGrad(float *P, float *Pu_f, float *grad, int X, int Z, int T, int t)
{
    //coordenadas no bloco
    const int x = threadIdx.x;
    const int z = threadIdx.y;

    //coordenadas em Ps (shared)
    const int x_s = threadIdx.x + prec_deriv;
    const int z_s = threadIdx.y + prec_deriv;

    //coordenadas da origem do bloco
    const int x_b = blockIdx.x * blockDim.x;
    const int z_b = blockIdx.y * blockDim.y;

    //coordenadas em P (global)
    const int x_g = x_b + x;
    const int z_g = z_b + z;

    const int shared_width = BLOCK_SIZE + 2*prec_deriv;
    const int tam_shared = (shared_width)*(shared_width);

    //nao vale a pena colocar cquad e P(t-2) na memoria shared de acordo com os testes
    __shared__ float Ps[shared_width][shared_width];


    if(z==0) //first warp in block
    {
	//copy P to shared memory
	for(int id=x; id<tam_shared; id+=WARP_SIZE)
	{
	    //coordenada dentro de Ps (shared) sendo lida de P
	    const int xx = id/shared_width;
	    const int zz = id%shared_width;

	    //coordenada correspondente na memoria global
	    const int x_c = x_b - prec_deriv + xx;
	    const int z_c = z_b - prec_deriv + zz;

	    //retirar if adicionando o anel de zeros
	    if(inbounds(x_c, z_c, prec_deriv))
		Ps[zz][xx] = P[xzt(x_c, z_c, t-1)];
	    else
		Ps[zz][xx] = 0.0f;
	}
    }
    __syncthreads();

    float lapP = deriv_x(Ps, x_s, z_s) + deriv_z(Ps, x_s, z_s);
    grad[xzt2(x_g, z_g, 0)] += Pu_f[xzt2(x_g,z_g,0) + X*Z*(T-1-t)] * lapP;

}

void
allocate_mem_simulate()
{
    hipMalloc(&P, X * Z * 4 * sizeof (float));	//pressao direta
    hipMalloc(&initial, X * Z * 2 * sizeof (float));	//pressao direta
    hipMalloc(&cquad, X * Z * sizeof (float));	//campo de velocidades
    hipMalloc(&source, T * n_source * sizeof (float));	//termos de fonte
    hipMalloc(&record_buffer, T * n_sensor * sizeof (float));	//buffer dos sensores

    hipMalloc(&pos_source_x, n_source*sizeof(int)); //posicoes das fontes
    hipMalloc(&pos_source_z, n_source*sizeof(int)); //posicoes das fontes
    hipMalloc(&pos_sensor_x, n_sensor*sizeof(int)); //posicoes dos sensores
    hipMalloc(&pos_sensor_z, n_sensor*sizeof(int)); //posicoes dos sensores 

    hipHostMalloc(&recording_h, T*n_sensor*sizeof(float));
    //recording_h = (float*)malloc(T*n_sensor*sizeof(float));

    allocated = 1;
}

void
free_mem_simulate()
{
    hipFree (P);
    hipFree (initial);
    hipFree (cquad);
    hipFree (source);
    hipFree (record_buffer);

    hipFree(pos_source_x);
    hipFree(pos_source_z);
    hipFree(pos_sensor_x);
    hipFree(pos_sensor_z);

    hipFree(recording_h);

    allocated = 0;
}

extern "C" void
init_memory_sim(int x, int z, int t, float *cq, 
	int ns, int *ps_x, int *ps_z, int nm, int *pm_x, int *pm_z, 
	float *src, float *init, float **rec)
{
    X = x; 
    Z = z; 
    T = t; 
    n_source = ns; 
    n_sensor = nm;

    if(allocated)
	free_mem_simulate();

    hipDeviceReset();

    allocate_mem_simulate();

    n_blocksX = X/BLOCK_SIZE;
    n_blocksZ = Z/BLOCK_SIZE;
    n_blocksS = n_sensor/(BLOCK_SIZE*BLOCK_SIZE);
    n_blocksF = n_source/(BLOCK_SIZE*BLOCK_SIZE);

    if(n_blocksX*BLOCK_SIZE != X)
	n_blocksX++;
    if(n_blocksZ*BLOCK_SIZE != Z)
	n_blocksZ++;
    if(n_blocksS*(BLOCK_SIZE*BLOCK_SIZE) != n_sensor) 
	n_blocksS++;
    if(n_blocksF*(BLOCK_SIZE*BLOCK_SIZE) != n_source) 
	n_blocksF++;


    blockGrid = {n_blocksX, n_blocksZ, 1};


    *rec = recording_h;
    hipMalloc(&P, X * Z * 4 * sizeof (float));	//pressao direta
    hipMemcpy(cquad, cq, X * Z * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(source, src, T * n_source * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(pos_sensor_x, pm_x, n_sensor*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_sensor_z, pm_z, n_sensor*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_source_x, ps_x, n_source*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_source_z, ps_z, n_source*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(initial, init, X*Z*2*sizeof(float), hipMemcpyHostToDevice);
}


void
allocate_mem_regress()
{
    hipMalloc(&observed, T * n_sensor * sizeof (float));	//buffer dos sensores
    hipMalloc(&grad, X*Z*sizeof(float));
    hipMalloc(&adj_source, T*n_sensor*sizeof(float));

    hipMalloc(&P_ub, X * Z * 4 * sizeof (float));	//pressao direta
    hipMalloc(&P_uf, X * Z * 4 * sizeof (float));	//pressao direta
    hipMalloc(&P_uf_full, X*Z*T*sizeof(float)); //campo adjoint completo

    hipHostMalloc(&simulated_h, T*n_sensor*sizeof(float));
    hipHostMalloc(&adj_source_h, T*n_sensor*sizeof(float));
    hipHostMalloc(&grad_h, X*Z*sizeof(float));

    allocated = 1;
}


void
free_mem_regress()
{
    hipFree (observed);
    hipFree (grad);
    hipFree(adj_source);

    hipFree (P_ub);
    hipFree (P_uf);
    hipFree (P_uf_full);

    hipFree(simulated_h);
    hipFree(adj_source_h);
    hipFree (grad_h);

    allocated = 0;
}

extern "C" void
init_memory_reg(float *obs, float **grad_ptr)
{

    allocate_mem_regress();

    hipMemcpy(observed, obs, T * n_source * sizeof (float), hipMemcpyHostToDevice);
    *grad_ptr = grad_h;
}



extern "C" void
setCquad(float *cq)
{
    hipMemcpy(cquad, cq, X * Z * sizeof (float), hipMemcpyHostToDevice);
}


extern "C" void
set_source(int ns, int *sx, int *sz, float *src)
{
    n_source = ns;
    hipMemcpy(pos_sensor_x, sx, ns * sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(pos_sensor_z, sz, ns * sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(source, src, ns * T * sizeof (float), hipMemcpyHostToDevice);
}


extern "C" void
cuda_simulate2 (int en_out, int idx_source)
{
    FILE *pipeout;
    float *frame_buffer;
    if (en_out) 
    {
	char mpegCom[500];
	sprintf(mpegCom, "ffmpeg -y -f rawvideo -vcodec rawvideo -pix_fmt gray -s %ix%i -r 60 -i - -f mp4 -q:v 5 -an -vcodec h264 -crf 0 output/output1P.mp4 -nostats -loglevel quiet", X, Z);    
	pipeout = popen(mpegCom, "w");    
	hipHostMalloc(&frame_buffer, X*Z*sizeof(float));
    }

    //copia condicoes iniciais
    hipMemset(P, 0, X*Z*4*sizeof(float));
    hipMemcpy(P, initial, X*Z*2*sizeof(float), hipMemcpyDeviceToDevice);

    for (int t = 0; t < T; t++)
    {
	//primeiros 2 frames sao condicao de contorno, logo nao calculados
	if (t > 1)
	{
	    simulateFrame <<<blockGrid, threadGrid>>> (P, cquad, X, Z, t);
	    hipDeviceSynchronize ();

	    somaFonte<<<n_blocksF, threadGrid>>>(P, X, Z, T, t, pos_source_x, pos_source_z, source, n_source, 0, idx_source);
	    hipDeviceSynchronize ();
	}
		
	// grava resultado nos sensores
	//gravaBufferSensores2ordem<<<1, n_sensor>>>(P, record_buffer, X, Z, T, t, pos_sensor_x, pos_sensor_z);
	gravaBufferSensores2ordem<<<n_blocksS, threadGrid>>>(P, record_buffer, X, Z, T, t, pos_sensor_x, pos_sensor_z, n_sensor);
	hipDeviceSynchronize();

	if (en_out)
	{
	    hipMemcpy(frame_buffer, &P[xzt(0,0,t)], X*Z*sizeof(float), hipMemcpyDeviceToHost);
	    writeFramePipe (pipeout, frame_buffer, X, Z, t, pos_sensor_x, pos_sensor_z, n_sensor);
	}
    }

    if(en_out)
    {
	fflush(pipeout);
	pclose(pipeout);
	hipFree(frame_buffer);
    }
    
    hipMemcpy (recording_h, record_buffer, T * n_sensor * sizeof (float), hipMemcpyDeviceToHost);
}


extern "C" void
cuda_grad_ext (float *mse_h, adj_func adj_calc, int idx_source)
{
    float mse;

    //zera campos de pressao
    hipMemset(P, 0, X*Z*4*sizeof(float));
    hipMemset(P_ub, 0, X*Z*4*sizeof(float));
    hipMemset(P_uf, 0, X*Z*4*sizeof(float));
    hipMemset(grad, 0, X*Z*sizeof(float));


    //FASE 1 - SIMULAÇÃO DIRETA    
    mse = 0;
    for (int t = 0; t < T; t++)
    {
	//primeiros 2 frames sao condicao de contorno, logo nao calculados
	if (t > 1)
	{
	    simulateFrame <<<blockGrid, threadGrid>>> (P, cquad, X, Z, t);
	    hipDeviceSynchronize ();

	    somaFonte<<<n_blocksF, threadGrid>>>(P, X, Z, T, t, pos_source_x, pos_source_z, source, n_source, 0, idx_source);
	    hipDeviceSynchronize ();
	}

	//grava resultado nos sensores
	gravaBufferSensores2ordem<<<n_blocksS, threadGrid>>>(P, record_buffer, X, Z, T, t, pos_sensor_x, pos_sensor_z, n_sensor);
	hipDeviceSynchronize ();

	    //hipMemcpy(frame_buffer, &P[xzt(0,0,t)], X*Z*sizeof(float), hipMemcpyDeviceToHost);
	    //writeFramePipe (pipeout, frame_buffer, X, Z, t, pos_sensor_x, pos_sensor_z, n_sensor);
    }


	//calcula MSE
	//for(int j=0;j<n_sensor;j++)
//	   {
//	   float resid = P[xzt(pos_sensor_x[j], pos_sensor_z[j], t)] - observed[j*T + t];
//	   mse += resid*resid;
//	   } 


    //FASE 1A - CÁLCULO DO MSE E ADJOINT SOURCE
    hipMemcpy(simulated_h, record_buffer, T*n_sensor*sizeof(float), hipMemcpyDeviceToHost);
    adj_calc (simulated_h, adj_source_h);
    hipMemcpy(adj_source, adj_source_h, T*n_sensor*sizeof(float), hipMemcpyHostToDevice);


    //FASE 2 - SIMULAÇÃO ADJOINT FORWARD
    int last_frame = 0;
    for (int t = 0; t < T; t++)
    {
	//primeiros 2 frames sao condicao de contorno, logo nao calculados
	if (t > 1)
	{
	    //-4 pq nao processa as bordas e laplaciano é de 5 pontos (margem 2)
	    simulateFrame <<<blockGrid, threadGrid>>> (P_uf, cquad, X, Z, t);
	    hipDeviceSynchronize ();

	    //soma termo de fonte - residuo flipado no tempo
	    somaFonte<<<n_blocksS, threadGrid>>>(P_uf, X, Z, T, t, pos_sensor_x, pos_sensor_z, adj_source, n_sensor, 1);
	    hipDeviceSynchronize ();
	}

	// TODO
	hipMemcpy(&P_uf_full[X*Z*t], &P_uf[xzt(0,0,t)], X*Z*sizeof(float), hipMemcpyDeviceToDevice);
	last_frame = t;

    }

    //FASE 3 - SIMULAÇÃO ADJOINT BACKWARD + SIMULAÇÃO DIRETA

    //zera campo P
    hipMemset(P, 0, X*Z*4*sizeof(float));

    //copia 2 ultimos frames da simulacao forward como condicao inicial da backward
    hipMemcpy(P_ub, &P_uf[xzt(0,0,last_frame)],  X*Z*sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(&P_ub[X*Z], &P_uf[xzt(0,0,last_frame-1)],  X*Z*sizeof(float), hipMemcpyDeviceToDevice);


    for (int t = 0; t < T; t++)
    {
	//primeiros 2 frames sao condicao de contorno, logo nao calculados
	if (t > 1)
	{
	    //-4 pq nao processa as bordas e laplaciano é de 5 pontos (margem 2)
	    simulateFrame <<<blockGrid, threadGrid>>> (P_ub, cquad, X, Z, t);
	    simulateFrame <<<blockGrid, threadGrid>>> (P, cquad, X, Z, t);
	    hipDeviceSynchronize ();

	    //soma termo de fonte
	    somaFonte<<<n_blocksF, threadGrid>>>(P, X, Z, T, t, pos_source_x, pos_source_z, source, n_source, 0, idx_source);
	    somaFonte<<<n_blocksS, threadGrid>>>(P_ub, X, Z, T, t, pos_sensor_x, pos_sensor_z, adj_source, n_sensor, 0);
	    hipDeviceSynchronize ();
	}
	//hipMemcpy(frame_buffer, &P[xzt(0,0,t)], X*Z*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(frame_buffer, &P_uf_full[X*Z*(T-1 - t)], X*Z*sizeof(float), hipMemcpyDeviceToHost);
	//writeFramePipe (pipeout, frame_buffer, X, Z, t, pos_sensor_x, pos_sensor_z, n_sensor);

	//trocar pela derivada no tempo? a principio muito mais rapido
	calcGrad<<<blockGrid, threadGrid>>> (P, P_uf_full, grad, X, Z, T, t);
	hipDeviceSynchronize();
    }

    hipMemcpy (grad_h, grad, X * Z * sizeof (float), hipMemcpyDeviceToHost);
    *mse_h = mse;

}


